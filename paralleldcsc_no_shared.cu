/* 
 * 
 * Jason Yik
 * jyik@usc.edu
 * EE451 Final Project
 *
 * CUDA implementation of DCSC matrix multiplication.
 * Implementation without shared memory.
 * 
 * Run on USC HPC:
 * srun -n1 --gres=gpu:1 ./parallel <n> <nnz>
 *
 * Run with executable in a graphs folder containing n_nnz_x graphs
 *
 */ 


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define GRID_WIDTH 128
#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 32

//TODO: possibly increase block width, decrease block height since there will be max 8 nonzeroes per col on avg

typedef struct {
	char column;
	int *JC;
	int *IR;
	int *NUM;

	int n;   // number of columns
	int nzc; // non zero columns/rows
	int nnz; // number of nonzeroes
} cs_matrix_t;

typedef struct {
	char column;
	int *JC;
	int *CP;
	int *IR;
	int *NUM;

	int n;   // number of columns
	int nzc; // non zero columns/rows
	int nnz; // number of nonzeroes
} dcs_matrix_t;

//uses CUDA unified memory
void cuda_cs(cs_matrix_t *m, const char *file, int n, int nnz, char column, int random_seed) {
	m->column = column;
	m->n = n;

	FILE *fp = fopen(file, "r");
	int bufc, bufr;

	hipMallocManaged(&(m->JC), sizeof(int) * (n+1));
	hipMallocManaged(&(m->IR), sizeof(int) * nnz);
	hipMallocManaged(&(m->NUM), sizeof(int) * nnz);
	
	srand(random_seed);
	//srand(time(0));

	int nzc = 0;

	int i = 0;
	int current_index = 0;
	fscanf(fp, "%d", &bufc);
	while(!feof(fp)) {
		//if blank columns, fill in JC
		while(i < bufc) {
			m->JC[i] = current_index;
			i++;
		}

		m->JC[i] = current_index;
		nzc++;
		while(!feof(fp) && i == bufc) {
			fscanf(fp, "%d", &bufr);
			m->IR[current_index] = bufr;
			m->NUM[current_index] = rand() % 100; 
			current_index++;
			
			fscanf(fp, "%d", &bufc);
		}
		i++;
	}
	//fill in the remainder of JC
	while(i <= n) {
		//current_index should now be greater than size of IR/NUM
		m->JC[i] = current_index; 
		i++;
	}

	m->nnz = current_index;
	m->nzc = nzc;

	fclose(fp);
	return;
}

//uses CUDA unified memory
void cuda_dcs(cs_matrix_t *m, dcs_matrix_t *d) {
	d->column = m->column;
	d->IR = m->IR;
	d->NUM = m->NUM;

	d->n = m->n;
	d->nnz = m->nnz;
	d->nzc = m->nzc;

	hipMallocManaged(&(d->JC), sizeof(int) * m->nzc);
	hipMallocManaged(&(d->CP), sizeof(int) * (m->nzc +1));

	int current_index = 0;
	for(int i = 0; i < m->n; i++) {
		if(m->JC[i] == m->JC[i+1]) {
			continue;
		}
		else {
			d->JC[current_index] = i;
			d->CP[current_index] = m->JC[i];
			current_index++;
		}
	}
	d->CP[current_index] = m->nnz;

	//invalidate m
	hipFree(m->JC);

	return;
}

__device__ int binary_search(int *arr, int len, int target) {
	int left = 0;
	int right = len -1;
	int mid;

	while(left <= right) {
		mid = (left + right)/2;
		if(arr[mid] == target) return mid;
		if(arr[mid] < target) {
			left = mid +1;
			continue;
		}
		if(arr[mid] > target) {
			right = mid -1;
			continue;
		}
	}

	return -1;
}

__global__ void device_multiply(dcs_matrix_t A, dcs_matrix_t B, int *C, int num_cols_per_block, int n) {
	int block_first = blockIdx.x * num_cols_per_block;
	if(block_first > B.nzc) return; // more blocks than nzc
	int block_last = block_first + num_cols_per_block; //exclusive
	if(block_last > B.nzc) block_last = B.nzc;

	//TODO: setup shared memory --> while loop for each thread, once all threads break they can sync

	//loop for the columns that this will look at
	int x = block_first + threadIdx.x; // index in B.JC this thread col is working on
	while(x < block_last) {
		int j = B.JC[x];
		int first = B.CP[x];
		int last = B.CP[x+1];
		int curr = first + threadIdx.y; // row index in B.IR this thread is working on
		//loop for the nonzero elements that this thread will execute on
		while(curr < last) {
			//do the multiplication, remember to atomicAdd for C
			int brow = B.IR[curr];
			int bval = B.NUM[curr];

			int apos = binary_search(A.JC, A.nzc, brow);
			if(apos != -1) {
				int acurr = A.CP[apos];
				int alast = A.CP[apos+1];

				int i, aval;
				while(acurr != alast) { // iterate over elements in column brow of A
					i = A.IR[acurr];
					aval = A.NUM[acurr];

					// C[i * n + j] += aval * bval;
					atomicAdd(C + (i*n + j), aval * bval); // race conditions may occur within this thread row

					acurr++;
				}
			}

			curr += blockDim.y; // next non-zero assigned round robin
		}
		
		x += blockDim.x; // next column is assigned round robin
	}

	//note: threads don't have to wait for each other to sync, some can be on different columns than others no problem
}

void parallel_multiply(int *C, int n, int nnz, char *Afile, char *Bfile, int Arseed, int Brseed) {
	//start timer
	struct timespec start, computation_done;
	double time;
	if( clock_gettime(CLOCK_REALTIME, &start) == -1) { perror("clock gettime");}

	//setup
	cs_matrix_t m1;
	dcs_matrix_t A;
	cuda_cs(&m1, Afile, n, nnz, 1, Arseed);
	cuda_dcs(&m1, &A);

	cs_matrix_t m2;
	dcs_matrix_t B;
	cuda_cs(&m2, Bfile, n, nnz, 1, Brseed);
	cuda_dcs(&m2, &B);

	//call device multiply
	dim3 dimGrid(GRID_WIDTH);
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);

	double nd = (double)B.nzc / (double)GRID_WIDTH;
	int num_cols_per_block = (int) ceil(nd);

	device_multiply<<<dimGrid, dimBlock>>>(A, B, C, num_cols_per_block, n);
	hipDeviceSynchronize(); // in order to access unified memory

	//stop timer for computation
	if( clock_gettime(CLOCK_REALTIME, &computation_done) == -1) { perror("clock gettime");}

	//print out time for finishing computation and copying back data
	time = (computation_done.tv_sec - start.tv_sec)+ (double)(computation_done.tv_nsec - start.tv_nsec)/1e9;
	printf("DCSC A nnz: %d, A nzc: %d, B nnz: %d, B nzc: %d\nExecution Time: %f\n", A.nnz, A.nzc, B.nnz, B.nzc, time);

	//free unified memory
	hipFree(A.JC);
	hipFree(A.CP);
	hipFree(A.IR);
	hipFree(A.NUM);

	hipFree(B.JC);
	hipFree(B.CP);
	hipFree(B.IR);
	hipFree(B.NUM);
}

int main(int argc, char **argv) {
	if(argc < 3) {
		printf("arguments: n nnz\n");
		return 1;
	}
	
	int n = atoi(argv[1]);
	int nnz = atoi(argv[2]);
	int num_iterations = 5;

	int *C;
	char Afile[20];
	char Bfile[20];
	hipMallocManaged(&C, sizeof(int)*n*n);

	//number of iterations of the program
	for(int it = 0; it < num_iterations; it++) {
		//initialize C
		for(int i = 0; i < n*n; i++) {
			C[i] = 0;
		}

		sprintf(Afile, "%d_%d_%d", n, nnz, it);
		sprintf(Bfile, "%d_%d_%d", n, nnz, (it+1) % num_iterations);

		//execute multiplication
		parallel_multiply(C, n, nnz, Afile, Bfile, 1, 1);

		//verify that C is correct here - deleted for sake of execution time
	}

	//TODO: since submitting CUDA takes so long on HPC, probabaly want to write batch
	// or run all of the tests in the same executable sequence, make changes after the
	// algorithm is shown to work
	
	hipFree(C);

	return 0;
}
