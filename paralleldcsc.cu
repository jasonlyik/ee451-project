#include "hip/hip_runtime.h"
/* 
 * 
 * Jason Yik
 * jyik@usc.edu
 * EE451 Final Project
 *
 * CUDA implementation of DCSC matrix multiplication.
 * 
 * Run on USC HPC:
 * srun -n1 --gres=gpu:1 ./parallel <n> <nnz>
 *
 * Run with executable in a graphs folder containing n_nnz_x graphs
 *
 */ 

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hipblas.h>

#define GRID_WIDTH 128
#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 32

//TODO: delete this
void read_matrix(int **m, char *file, int seed) {
	FILE *fp = fopen(file, "r");

	int bufc, bufr;
	srand(seed);

	fscanf(fp, "%d", &bufc);
	while(!feof(fp)) {
		fscanf(fp, "%d", &bufr);
		m[bufr][bufc] = rand() % 100;
		fscanf(fp, "%d", &bufc);
	}

	fclose(fp);
	return;
}

//TODO: delete this
void serial_multiply(int **C, int n, char *Afile, char *Bfile, int Arseed, int Brseed) {
	//O(n^3) block multiplication from pa1
	struct timespec start, stop;
	if( clock_gettime(CLOCK_REALTIME, &start) == -1) { perror("clock gettime");}

	//read graph into matrices
	int **A = (int **) malloc (sizeof(int *)*n);
	int **B = (int **) malloc (sizeof(int *)*n);
	for (int i=0; i<n; i++) {
		A[i] = (int *) malloc(sizeof(int)*n);
		B[i] = (int *) malloc(sizeof(int)*n);
	}

	for(int i = 0; i < n; i++) {
		for(int j = 0; j < n; j++) {
			A[i][j] = 0;
			B[i][j] = 0;
		}
	}

	read_matrix(A, Afile, Arseed);
	read_matrix(B, Bfile, Brseed);

	int b = 16;
	int m = n / b;
	int i, j;
	int k, u, v, w;
	for(i = 0; i < m; i++) {
		for(j = 0; j < m; j++) {
			//iterate over every block in A's row, B's column
			for(k = 0; k < m; k++) {
				//multiply A block by B block
				for(u = 0; u < b; u++) {
					for(v = 0; v < b; v++) {
						for(w = 0; w < b; w++) {
							//C(i, j)(u, v) += A(i, k)(u, w) * B(k, j)(w, v)
							C[i*b + u][j*b + v] = C[i*b + u][j*b + v] + A[i*b + u][k*b + w] * B[k*b + w][j*b + v];
						}
					}
				}
			}
		}
	}

	if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror("clock gettime");}	

	//print time
	double time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
	printf("Serial Block Multiplication\nExecution time: %f\n", time);

	for (i=0; i<n; i++) {
		free(A[i]);
		free(B[i]);
	}
	free(A);
	free(B);

	return;
}

//uses CUDA unified memory
void cuda_cs(cs_matrix_t *m, const char *file, int n, int nnz, char column, int random_seed) {
	m->column = column;
	m->n = n;

	FILE *fp = fopen(file, "r");
	int bufc, bufr;

	hipMallocManaged(&(m->JC), sizeof(int) * (n+1));
	hipMallocManaged(&(m->IR), sizeof(int) * nnz);
	hipMallocManaged(&(m->NUM), sizeof(int) * nnz);
	
	srand(random_seed);
	//srand(time(0));

	int nzc = 0;

	int i = 0;
	int current_index = 0;
	fscanf(fp, "%d", &bufc);
	while(!feof(fp)) {
		//if blank columns, fill in JC
		while(i < bufc) {
			m->JC[i] = current_index;
			i++;
		}

		m->JC[i] = current_index;
		nzc++;
		while(!feof(fp) && i == bufc) {
			fscanf(fp, "%d", &bufr);
			m->IR[current_index] = bufr;
			m->NUM[current_index] = rand() % 100; 
			current_index++;
			
			fscanf(fp, "%d", &bufc);
		}
		i++;
	}
	//fill in the remainder of JC
	while(i <= n) {
		//current_index should now be greater than size of IR/NUM
		m->JC[i] = current_index; 
		i++;
	}

	m->nnz = current_index;
	m->nzc = nzc;

	fclose(fp);
	return;
}

//uses CUDA unified memory
void cuda_dcs(cs_matrix_t *m, dcs_matrix_t *d) {
	d->column = m->column;
	d->IR = m->IR;
	d->NUM = m->NUM;

	d->n = m->n;
	d->nnz = m->nnz;
	d->nzc = m->nzc;

	hipMallocManaged(&(d->JC), sizeof(int) * m->nzc);
	hipMallocManaged(&(d->CP), sizeof(int) * (m->nzc +1));

	int current_index = 0;
	for(int i = 0; i < m->n; i++) {
		if(m->JC[i] == m->JC[i+1]) {
			continue;
		}
		else {
			d->JC[current_index] = i;
			d->CP[current_index] = m->JC[i];
			current_index++;
		}
	}
	d->CP[current_index] = m->nnz;

	//invalidate m
	hipFree(m->JC);

	return;
}

__device__ int binary_search(int *arr, int len, int target) {
	int left = 0;
	int right = len -1;
	int mid;

	while(left <= right) {
		mid = (left + right)/2;
		if(arr[mid] == target) return mid;
		if(arr[mid] < target) {
			left = mid +1;
			continue;
		}
		if(arr[mid] > target) {
			right = mid -1;
			continue;
		}
	}

	return -1;
}

__global__ void device_multiply(dcs_matrix_t A, dcs_matrix_t B, int *C, int n) {
	//TODO: write this
	
}

void parallel_multiply(int *C, int n, int nnz, char *Afile, char *Bfile, int Arseed, int Brseed) {
	//start timer
	struct timespec start, computation_done;
	if( clock_gettime(CLOCK_REALTIME, &start) == -1) { perror("clock gettime");}

	//setup
	cs_matrix_t m1;
	dcs_matrix_t A;
	cuda_cs(&m1, Afile, n, nnz, 1, Arseed);
	cuda_dcs(&m1, &A);

	cs_matrix_t m2;
	dcs_matrix_t B;
	cuda_cs(&m2, Bfile, n, nnz, 1, Brseed);
	cuda_dcs(&m2, &B);

	//call device multiply
	dim3 dimGrid(GRID_WIDTH);
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);

	device_multiply<<<dimGrid, dimBlock>>>(A, B, C, n);
	hipDeviceSynchronize(); // in order to access unified memory

	//stop timer for computation
	if( clock_gettime(CLOCK_REALTIME, &computation_done) == -1) { perror("clock gettime");}

	//print out time for finishing computation and copying back data
	time = (computation_done.tv_sec - start.tv_sec)+ (double)(computation_done.tv_nsec - start.tv_nsec)/1e9;
	printf("DCSC A nnz: %d, A nzc: %d, B nnz: %d, B nzc: %d\nExecution Time: %f\n", A.nnz, A.nzc, B.nnz, B.nzc, time);

	//free unified memory
	hipFree(A.JC);
	hipFree(A.CP);
	hipFree(A.IR);
	hipFree(A.NUM);

	hipFree(B.JC);
	hipFree(B.CP);
	hipFree(B.IR);
	hipFree(B.NUM);
}

int main(int argc, char **argv) {
	if(argc < 3) {
		printf("arguments: n nnz\n");
		return 1;
	}
	
	int n = atoi(argv[1]);
	int nnz = atoi(argv[2]);
	int num_iterations = 5;

	hipMallocManaged(&C, sizeof(int)*n*n);

	//number of iterations of the program
	for(int it = 0; it < num_iterations; it++) {
		//initialize C
		for(int i = 0; i < n^2; i++) {
			C[i] = 0;
		}

		sprintf(Afile, "%d_%d_%d", n, nnz, it);
		sprintf(Bfile, "%d_%d_%d", n, nnz, (it+1) % num_iterations);

		//execute multiplication
		parallel_multiply(C, n, nnz, Afile, Bfile, 1, 1);

		//verify that C is correct here
		//TODO: get rid of this
		if(it == 0) {
			int **C2 = (int **) malloc (sizeof(int *)*n);
			for (int i=0; i<n; i++) {
				C2[i] = (int *) malloc(sizeof(int)*n);
			}
			for(int i = 0; i < n; i++) {
				for(int j = 0; j < n; j++) {
					C2[i][j] = 0;
				}
			}
			serial_multiply(C2, n, Afile, Bfile, 1, 1);

			char correct = 1;
			for(int i = 0; i < n; i++) {
				for(int j = 0; j < n; j++) {
					if(C[i*n + j] != C2[i][j]) {
						correct = 0;
						printf("Output Matrices differ at [%d][%d]\n", i, j);
					}
				}
			}
			if(correct) printf("Output Matrices do not differ\n");

			for (int i=0; i<n; i++) {
				free(C2[i]);
			}
			free(C2);
		}

	}
	
	hipFree(C);

	return 0;
}