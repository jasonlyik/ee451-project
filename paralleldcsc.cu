/* 
 * 
 * Jason Yik
 * jyik@usc.edu
 * EE451 Final Project
 *
 * CUDA implementation of DCSC matrix multiplication.
 * 
 * Run on USC HPC:
 * srun -n1 --gres=gpu:1 ./parallel <n> <nnz>
 *
 * Run with executable in a graphs folder containing n_nnz_x graphs
 *
 */ 


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define GRID_WIDTH 128
#define BLOCK_WIDTH 32
#define BLOCK_HEIGHT 8


typedef struct {
	char column;
	int *JC;
	int *IR;
	int *NUM;

	int n;   // number of columns
	int nzc; // non zero columns/rows
	int nnz; // number of nonzeroes
} cs_matrix_t;

typedef struct {
	char column;
	int *JC;
	int *CP;
	int *IR;
	int *NUM;

	int n;   // number of columns
	int nzc; // non zero columns/rows
	int nnz; // number of nonzeroes
} dcs_matrix_t;


void read_matrix(int **m, char *file, int seed) {
	FILE *fp = fopen(file, "r");

	int bufc, bufr;
	srand(seed);

	fscanf(fp, "%d", &bufc);
	while(!feof(fp)) {
		fscanf(fp, "%d", &bufr);
		m[bufr][bufc] = rand() % 100;
		fscanf(fp, "%d", &bufc);
	}

	fclose(fp);
	return;
}

void serial_multiply(int **C, int n, char *Afile, char *Bfile, int Arseed, int Brseed) {
	//O(n^3) block multiplication from pa1
	struct timespec start, stop;
	if( clock_gettime(CLOCK_REALTIME, &start) == -1) { perror("clock gettime");}

	//read graph into matrices
	int **A = (int **) malloc (sizeof(int *)*n);
	int **B = (int **) malloc (sizeof(int *)*n);
	for (int i=0; i<n; i++) {
		A[i] = (int *) malloc(sizeof(int)*n);
		B[i] = (int *) malloc(sizeof(int)*n);
	}

	for(int i = 0; i < n; i++) {
		for(int j = 0; j < n; j++) {
			A[i][j] = 0;
			B[i][j] = 0;
		}
	}

	read_matrix(A, Afile, Arseed);
	read_matrix(B, Bfile, Brseed);

	int b = 16;
	int m = n / b;
	int i, j;
	int k, u, v, w;
	for(i = 0; i < m; i++) {
		for(j = 0; j < m; j++) {
			//iterate over every block in A's row, B's column
			for(k = 0; k < m; k++) {
				//multiply A block by B block
				for(u = 0; u < b; u++) {
					for(v = 0; v < b; v++) {
						for(w = 0; w < b; w++) {
							//C(i, j)(u, v) += A(i, k)(u, w) * B(k, j)(w, v)
							C[i*b + u][j*b + v] = C[i*b + u][j*b + v] + A[i*b + u][k*b + w] * B[k*b + w][j*b + v];
						}
					}
				}
			}
		}
	}

	if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror("clock gettime");}	

	//print time
	double time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
	printf("Serial Block Multiplication\nExecution time: %f\n", time);

	for (i=0; i<n; i++) {
		free(A[i]);
		free(B[i]);
	}
	free(A);
	free(B);

	return;
}

//uses CUDA unified memory
void cuda_cs(cs_matrix_t *m, const char *file, int n, int nnz, char column, int random_seed) {
	m->column = column;
	m->n = n;

	FILE *fp = fopen(file, "r");
	int bufc, bufr;

	hipMallocManaged(&(m->JC), sizeof(int) * (n+1));
	hipMallocManaged(&(m->IR), sizeof(int) * nnz);
	hipMallocManaged(&(m->NUM), sizeof(int) * nnz);
	
	srand(random_seed);
	//srand(time(0));

	int nzc = 0;

	int i = 0;
	int current_index = 0;
	fscanf(fp, "%d", &bufc);
	while(!feof(fp)) {
		//if blank columns, fill in JC
		while(i < bufc) {
			m->JC[i] = current_index;
			i++;
		}

		m->JC[i] = current_index;
		nzc++;
		while(!feof(fp) && i == bufc) {
			fscanf(fp, "%d", &bufr);
			m->IR[current_index] = bufr;
			m->NUM[current_index] = rand() % 100; 
			current_index++;
			
			fscanf(fp, "%d", &bufc);
		}
		i++;
	}
	//fill in the remainder of JC
	while(i <= n) {
		//current_index should now be greater than size of IR/NUM
		m->JC[i] = current_index; 
		i++;
	}

	m->nnz = current_index;
	m->nzc = nzc;

	fclose(fp);
	return;
}

//uses CUDA unified memory
void cuda_dcs(cs_matrix_t *m, dcs_matrix_t *d) {
	d->column = m->column;
	d->IR = m->IR;
	d->NUM = m->NUM;

	d->n = m->n;
	d->nnz = m->nnz;
	d->nzc = m->nzc;

	hipMallocManaged(&(d->JC), sizeof(int) * m->nzc);
	hipMallocManaged(&(d->CP), sizeof(int) * (m->nzc +1));

	int current_index = 0;
	for(int i = 0; i < m->n; i++) {
		if(m->JC[i] == m->JC[i+1]) {
			continue;
		}
		else {
			d->JC[current_index] = i;
			d->CP[current_index] = m->JC[i];
			current_index++;
		}
	}
	d->CP[current_index] = m->nnz;

	//invalidate m
	hipFree(m->JC);

	return;
}

__device__ int binary_search(int *arr, int len, int target) {
	int left = 0;
	int right = len -1;
	int mid;

	while(left <= right) {
		mid = (left + right)/2;
		if(arr[mid] == target) return mid;
		if(arr[mid] < target) {
			left = mid +1;
			continue;
		}
		if(arr[mid] > target) {
			right = mid -1;
			continue;
		}
	}

	return -1;
}

__global__ void device_multiply(dcs_matrix_t A, dcs_matrix_t B, int *C, int num_cols_per_block, int n) {
	int block_first = blockIdx.x * num_cols_per_block;
	if(block_first > B.nzc) return; // more blocks than nzc
	int block_last = block_first + num_cols_per_block; //exclusive
	if(block_last > B.nzc) block_last = B.nzc;

	int a_nzc = A.nzc;

	extern __shared__ int s[];
	int *a_jc = s; // length a_nzc
	// int *b_jc = &a_jc[a_nzc]; // length num_cols_per_block
	// int *b_cp = &b_jc[num_cols_per_block]; // length num_cols_per_block +1

	// copy A.JC into shared memory since always doing binary search on it
	int t_idx = threadIdx.x + threadIdx.y * blockDim.x;
	int tot_threads = blockDim.x * blockDim.y;
	int buf;

	buf = t_idx;
	while(buf < a_nzc) {
		a_jc[buf] = A.JC[buf];
		buf += tot_threads;
	}

	// // copy relevant portions of B.JC and B.CP into shared memory
	// int cols = block_last - block_first;
	// buf = t_idx; // index of b_jc
	// while(buf < cols) {
	// 	b_jc[buf] = B.JC[block_first + buf];
	// 	buf += tot_threads;
	// }
	// buf = t_idx; // index of b_cp
	// while(buf <= cols) {
	// 	b_cp[buf] = B.CP[block_first + buf];
	// 	buf += tot_threads;
	// }

	__syncthreads();

	
	int j, first, last, curr;
	int brow, bval, apos;
	int acurr, alast, i, aval;

	//loop for the columns that this will look at
	int x = block_first + threadIdx.x; // index in B.JC this thread col is working on
	while(x < block_last) {
		j = B.JC[x];
		first = B.CP[x];
		last = B.CP[x+1];
		curr = first + threadIdx.y; // row index in B.IR this thread is working on
		//loop for the nonzero elements that this thread will execute on
		while(curr < last) {
			//do the multiplication, remember to atomicAdd for C
			brow = B.IR[curr];
			bval = B.NUM[curr];

			apos = binary_search(a_jc, a_nzc, brow);
			if(apos != -1) {
				acurr = A.CP[apos];
				alast = A.CP[apos+1];

				while(acurr != alast) { // iterate over elements in column brow of A
					i = A.IR[acurr];
					aval = A.NUM[acurr];

					// C[i * n + j] += aval * bval;
					atomicAdd(C + (i*n + j), aval * bval); // race conditions may occur within this thread row

					acurr++;
				}
			}

			curr += blockDim.y; // next non-zero assigned round robin
		}
		
		x += blockDim.x; // next column is assigned round robin
	}

	//note: threads don't have to wait for each other to sync, some can be on different columns than others no problem
}

void parallel_multiply(int *C, int n, int nnz, char *Afile, char *Bfile, int Arseed, int Brseed) {
	//start timer
	struct timespec start, computation_done;
	double time;
	if( clock_gettime(CLOCK_REALTIME, &start) == -1) { perror("clock gettime");}

	//setup
	cs_matrix_t m1;
	dcs_matrix_t A;
	cuda_cs(&m1, Afile, n, nnz, 1, Arseed);
	cuda_dcs(&m1, &A);

	cs_matrix_t m2;
	dcs_matrix_t B;
	cuda_cs(&m2, Bfile, n, nnz, 1, Brseed);
	cuda_dcs(&m2, &B);

	//call device multiply
	dim3 dimGrid(GRID_WIDTH);
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);

	double nd = (double)B.nzc / (double)GRID_WIDTH;
	int num_cols_per_block = (int) ceil(nd);

	//dynamic allocation of shared memory
	//holds A.JC, B.JC, B.CP
	unsigned shared_space = sizeof(int) * (A.nzc);
	device_multiply<<<dimGrid, dimBlock, shared_space>>>(A, B, C, num_cols_per_block, n);
	hipDeviceSynchronize(); // in order to access unified memory

	//stop timer for computation
	if( clock_gettime(CLOCK_REALTIME, &computation_done) == -1) { perror("clock gettime");}

	//print out time for finishing computation and copying back data
	time = (computation_done.tv_sec - start.tv_sec)+ (double)(computation_done.tv_nsec - start.tv_nsec)/1e9;
	printf("DCSC A nnz: %d, A nzc: %d, B nnz: %d, B nzc: %d\nExecution Time: %f\n", A.nnz, A.nzc, B.nnz, B.nzc, time);

	//free unified memory
	hipFree(A.JC);
	hipFree(A.CP);
	hipFree(A.IR);
	hipFree(A.NUM);

	hipFree(B.JC);
	hipFree(B.CP);
	hipFree(B.IR);
	hipFree(B.NUM);
}

int main(int argc, char **argv) {
	if(argc < 3) {
		printf("arguments: n nnz\n");
		return 1;
	}
	
	char verify = 0; // CHANGE FOR NO VERIFICATION

	int n = atoi(argv[1]);
	int nnz = atoi(argv[2]);
	int num_iterations = 5;

	if(n > 2048) verify = 0; // override so serial compute doesn't take forever

	int *C = (int *) malloc (sizeof(int)*n*n);
	int *gpu_C;
	hipMalloc((void**)&gpu_C, sizeof(int)*n*n);

	char Afile[20];
	char Bfile[20];

	//number of iterations of the program
	for(int it = 0; it < num_iterations; it++) {
		//initialize C
		for(int i = 0; i < n*n; i++) {
			C[i] = 0;
		}
		hipMemcpy(gpu_C, C, sizeof(int)*n*n, hipMemcpyHostToDevice);

		sprintf(Afile, "%d_%d_%d", n, nnz, it);
		sprintf(Bfile, "%d_%d_%d", n, nnz, (it+1) % num_iterations);

		//execute multiplication
		parallel_multiply(gpu_C, n, nnz, Afile, Bfile, 1, 1);

		//verify that C is correct here
		if(it == 0 && verify) {
			hipMemcpy(C, gpu_C, sizeof(int)*n*n, hipMemcpyDeviceToHost);

			int **C2 = (int **) malloc (sizeof(int *)*n);
			for (int i=0; i<n; i++) {
				C2[i] = (int *) malloc(sizeof(int)*n);
			}
			for(int i = 0; i < n; i++) {
				for(int j = 0; j < n; j++) {
					C2[i][j] = 0;
				}
			}
			serial_multiply(C2, n, Afile, Bfile, 1, 1);

			char correct = 1;
			for(int i = 0; i < n; i++) {
				for(int j = 0; j < n; j++) {
					if(C[i*n + j] != C2[i][j]) {
						correct = 0;
						printf("Output Matrices differ at [%d][%d]\n", i, j);
						break;
					}
				}
				if(!correct) break;
			}

			for (int i=0; i<n; i++) {
				free(C2[i]);
			}
			free(C2);

			if(correct) {
				printf("Output Matrices do not differ\n");
			}
			else {
				printf("Incorrect Multiplication, stopping\n");
				break;
			}
		}

	}

	hipFree(gpu_C);
	free(C);

	return 0;
}